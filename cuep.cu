#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#include "cuep.h"

/* Constant (read only) device memory */
__constant__ double dev_startpos[3];
__constant__ double dev_endpos[3];
__constant__ double dev_beta[3];
__constant__ double dev_n;

__global__ void endpoint(int *dev_flag, double *dev_Em, double *dev_Ep, double *dev_x) {
    double r[3], rhat[3], R;
    double doppler, dot_product;

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < N) {
        dev_flag[tid] = 0;
    
        /* Calculate E_- */
        r[0] = dev_x[3*tid] - dev_startpos[0];
        r[1] = dev_x[3*tid+1] - dev_startpos[1];
        r[2] = dev_x[3*tid+2] - dev_startpos[2];
    
        R = sqrt(r[0]*r[0] + r[1]*r[1] + r[2]*r[2]);
    
        rhat[0] = r[0] / R;
        rhat[1] = r[1] / R;
        rhat[2] = r[2] / R;
    
        dot_product = dev_beta[0]*rhat[0] + dev_beta[1]*rhat[1] + dev_beta[2]*rhat[2];
        doppler = (1. - dev_n * dot_product)*R;
    
        if (doppler < SMALL_NUMBER) {
            dev_flag[tid] = 1;
            dev_Em[3*tid] = 0.0;
            dev_Em[3*tid+1] = 0.0;
            dev_Em[3*tid+2] = 0.0;
        }
        else {
            dev_Em[3*tid] = dot_product*rhat[0] - dev_beta[0];
            dev_Em[3*tid+1] = dot_product*rhat[1] - dev_beta[1];
            dev_Em[3*tid+2] = dot_product*rhat[2] - dev_beta[2];
        }
    
        /* Calculate E_+ */
        r[0] = dev_x[3*tid] - dev_endpos[0];
        r[1] = dev_x[3*tid+1] - dev_endpos[1];
        r[2] = dev_x[3*tid+2] - dev_endpos[2];
    
        R = sqrt(r[0]*r[0] + r[1]*r[1] + r[2]*r[2]);
    
        rhat[0] = r[0] / R;
        rhat[1] = r[1] / R;
        rhat[2] = r[2] / R;
    
        dot_product = dev_beta[0]*rhat[0] + dev_beta[1]*rhat[1] + dev_beta[2]*rhat[2];
        doppler = (1. - dev_n * dot_product)*R;
    
        if (doppler < SMALL_NUMBER) {
            dev_Ep[3*tid] = 0.0;
            dev_Ep[3*tid+1] = 0.0;
            dev_Ep[3*tid+2] = 0.0;
        }
        else {
            dev_flag[tid] = 0;
            dev_Ep[3*tid] = dot_product*rhat[0] - dev_beta[0];
            dev_Ep[3*tid+1] = dot_product*rhat[1] - dev_beta[1];
            dev_Ep[3*tid+2] = dot_product*rhat[2] - dev_beta[2];
        }

        tid += blockDim.x * gridDim.x;
    }
}

int cuep_create_plan(struct cuep_plan *d)
{
    hipError_t err;

    /* Allocate device memory */
    err = hipMalloc(&(d->dev_flag), N*sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMalloc(&(d->dev_Ep), 3*N*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMalloc(&(d->dev_Em), 3*N*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMalloc(&(d->dev_x), 3*N*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    return 0;
}

int cuep_execute_plan(double *Em, double *Ep, int *flag, double *x, double *startpos, double *endpos, double *beta, double n, struct cuep_plan *d)
{
    const int threadsPerBlock = MIN(N, MAX_THREADS_PER_BLOCK);
    const int numberOfBlocks = MIN((N+threadsPerBlock-1)/threadsPerBlock, MAX_BLOCKS);

    hipError_t err;
#ifdef TIME_CALL
    hipEvent_t start, stop;

    float elapsed_time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

    /* Copy to device */
    err = hipMemcpyToSymbol(HIP_SYMBOL(dev_startpos), startpos, 3*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(dev_endpos), endpos, 3*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(dev_beta), beta, 3*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(dev_n), &n, sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpy(d->dev_x, x, 3*N*sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    /* Call function on device */
#ifdef TIME_CALL
    hipEventRecord(start, 0);
#endif
    endpoint<<<numberOfBlocks, threadsPerBlock>>>(d->dev_flag, d->dev_Em, d->dev_Ep, d->dev_x);
#ifdef TIME_CALL
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    printf("launched %d blocks with %d threads each\n", (N+threadsPerBlock-1)/threadsPerBlock, threadsPerBlock);
    printf("Runtime: %.3f ms\n", elapsed_time);
#endif

    /* Copy result back to host */
    err = hipMemcpy(flag, d->dev_flag, N*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpy(Ep, d->dev_Ep, 3*N*sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpy(Em, d->dev_Em, 3*N*sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    return 0;
}

int cuep_destroy_plan(struct cuep_plan *d)
{
    hipError_t err;

    hipFree(d->dev_flag);
    hipFree(d->dev_Ep);
    hipFree(d->dev_Em);
    hipFree(d->dev_x);

    if ((err = hipGetLastError()) != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    return 0;
}

