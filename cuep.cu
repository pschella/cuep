#include <hip/hip_runtime.h>

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

/* Print runtime */
#define TIME_CALL

/* When the Doppler factor is to be considered zero */
#define SMALL_NUMBER 1.e-20

/* Number of antennas */
#define N 512

/* Scaling of random positions for antenna grid (arbitrary and not needed for actual code) */
#define SCALE 500

#define MIN(X, Y) ((X) < (Y) ? (X) : (Y))

/* The number of parallel GPU threads is determined by the product of the number of
 * thread blocks launched and the number of threads per block. There are hardware limits
 * on both numbers. If possible a thread is launched for each observer position.
 * If the number of observer positions exceeds the maximum number of threads allowed per block
 * (which is usually 1024 but is hardware dependent) than multiple blocks are launched.
 * The maximum number of threads launched in total is 1024 (blocks) * 1024 (threads per block)
 * if even more antennas are needed all threads loop over several antennas instead.
 * For performance one might want to play with the 1024 and try different powers of two (within hardware limits).
 */

#define MAX_BLOCKS 1024
#define MAX_THREADS_PER_BLOCK 1024

const int threadsPerBlock = MIN(N, MAX_THREADS_PER_BLOCK);
const int numberOfBlocks = MIN((N+threadsPerBlock-1)/threadsPerBlock, MAX_BLOCKS);

/* Constant (read only) device memory */
__constant__ double dev_startpos[3];
__constant__ double dev_endpos[3];
__constant__ double dev_beta[3];
__constant__ double dev_n;

/* Structure to store device memory pointers */
struct cuep_plan {
    int *dev_flag;
    double *dev_x;
    double *dev_Ep;
    double *dev_Em;
};

__global__ void endpoint(int *dev_flag, double *dev_Em, double *dev_Ep, double *dev_x) {
    double r[3], rhat[3], R;
    double doppler, dot_product;

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < N) {
        dev_flag[tid] = 0;
    
        /* Calculate E_- */
        r[0] = dev_x[3*tid] - dev_startpos[0];
        r[1] = dev_x[3*tid+1] - dev_startpos[1];
        r[2] = dev_x[3*tid+2] - dev_startpos[2];
    
        R = sqrt(r[0]*r[0] + r[1]*r[1] + r[2]*r[2]);
    
        rhat[0] = r[0] / R;
        rhat[1] = r[1] / R;
        rhat[2] = r[2] / R;
    
        dot_product = dev_beta[0]*rhat[0] + dev_beta[1]*rhat[1] + dev_beta[2]*rhat[2];
        doppler = (1. - dev_n * dot_product)*R;
    
        if (doppler < SMALL_NUMBER) {
            dev_flag[tid] = 1;
            dev_Em[3*tid] = 0.0;
            dev_Em[3*tid+1] = 0.0;
            dev_Em[3*tid+2] = 0.0;
        }
        else {
            dev_Em[3*tid] = dot_product*rhat[0] - dev_beta[0];
            dev_Em[3*tid+1] = dot_product*rhat[1] - dev_beta[1];
            dev_Em[3*tid+2] = dot_product*rhat[2] - dev_beta[2];
        }
    
        /* Calculate E_+ */
        r[0] = dev_x[3*tid] - dev_endpos[0];
        r[1] = dev_x[3*tid+1] - dev_endpos[1];
        r[2] = dev_x[3*tid+2] - dev_endpos[2];
    
        R = sqrt(r[0]*r[0] + r[1]*r[1] + r[2]*r[2]);
    
        rhat[0] = r[0] / R;
        rhat[1] = r[1] / R;
        rhat[2] = r[2] / R;
    
        dot_product = dev_beta[0]*rhat[0] + dev_beta[1]*rhat[1] + dev_beta[2]*rhat[2];
        doppler = (1. - dev_n * dot_product)*R;
    
        if (doppler < SMALL_NUMBER) {
            dev_Ep[3*tid] = 0.0;
            dev_Ep[3*tid+1] = 0.0;
            dev_Ep[3*tid+2] = 0.0;
        }
        else {
            dev_flag[tid] = 0;
            dev_Ep[3*tid] = dot_product*rhat[0] - dev_beta[0];
            dev_Ep[3*tid+1] = dot_product*rhat[1] - dev_beta[1];
            dev_Ep[3*tid+2] = dot_product*rhat[2] - dev_beta[2];
        }

        tid += blockDim.x * gridDim.x;
    }
}

int cuep_create_plan(struct cuep_plan *d)
{
    hipError_t err;

    /* Allocate device memory */
    err = hipMalloc(&(d->dev_flag), N*sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMalloc(&(d->dev_Ep), 3*N*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMalloc(&(d->dev_Em), 3*N*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMalloc(&(d->dev_x), 3*N*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    return 0;
}

int cuep_execute_plan(double *Em, double *Ep, int *flag, double *x, double *startpos, double *endpos, double *beta, double n, struct cuep_plan *d)
{
    hipError_t err;
#ifdef TIME_CALL
    hipEvent_t start, stop;

    float elapsed_time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

    /* Copy to device */
    err = hipMemcpyToSymbol(HIP_SYMBOL(dev_startpos), startpos, 3*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(dev_endpos), endpos, 3*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(dev_beta), beta, 3*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(dev_n), &n, sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpy(d->dev_x, x, 3*N*sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    /* Call function on device, launching 1 thread block with N threads */
#ifdef TIME_CALL
    hipEventRecord(start, 0);
#endif
    endpoint<<<numberOfBlocks, threadsPerBlock>>>(d->dev_flag, d->dev_Em, d->dev_Ep, d->dev_x);
#ifdef TIME_CALL
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    printf("launched %d blocks with %d threads each\n", (N+threadsPerBlock-1)/threadsPerBlock, threadsPerBlock);
    printf("Runtime: %.3f ms\n", elapsed_time);
#endif

    /* Copy result back to host */
    err = hipMemcpy(flag, d->dev_flag, N*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpy(Ep, d->dev_Ep, 3*N*sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpy(Em, d->dev_Em, 3*N*sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    return 0;
}

int cuep_destroy_plan(struct cuep_plan *d)
{
    hipError_t err;

    hipFree(d->dev_flag);
    hipFree(d->dev_Ep);
    hipFree(d->dev_Em);
    hipFree(d->dev_x);

    if ((err = hipGetLastError()) != hipSuccess) {
        fprintf(stderr, "Error %s\n", hipGetErrorString(err));
        return 1;
    }

    return 0;
}

int main(int argc, char* argv[])
{
    int i;
    int *flag;
    double startpos[3];
    double endpos[3];
    double beta[3];
    double n;
    double *x, *Ep, *Em;
    struct cuep_plan d;

    srand(time(NULL));

    n = 1.003;

    /* Allocate host memory */
    flag = (int*)malloc(N * sizeof(int));
    Ep = (double*)malloc(3 * N * sizeof(double));
    Em = (double*)malloc(3 * N * sizeof(double));
    x = (double*)malloc(3 * N * sizeof(double));

    /* Generate random particle */
    for (i=0; i<3; i++) {
        startpos[i] = 1000. * (double)rand() / RAND_MAX;
        endpos[i] = 1000. * (double)rand() / RAND_MAX;
        beta[i] = (double)rand() / RAND_MAX;
    }

    /* Generate random antenna positions */
    for (i=0; i<3*N; i++) {
        x[i] = SCALE * (double)rand() / RAND_MAX;
    }

    if (cuep_create_plan(&d) != 0) return 1;

    if (cuep_execute_plan(Em, Ep, flag, x, startpos, endpos, beta, n, &d) != 0) return 1;

    if (cuep_destroy_plan(&d) != 0) return 1;

    /* Print results */
    for (i=0; i<10; i++) {
        printf("flag %d E+ %.3f %.3f %.3f E- %.3f %.3f %.3f\n", flag[i], Ep[3*i], Ep[3*i+1], Ep[3*i+2], Em[3*i], Em[3*i+1], Em[3*i+2]);
    }

    /* Cleanup */
    free(flag);
    free(Ep);
    free(Em);
    free(x);
}

